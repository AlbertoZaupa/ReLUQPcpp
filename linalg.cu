#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "linalg.hpp"

// -----------------------------
// Matrix and Vector Classes
// -----------------------------

Matrix::Matrix() {
    rows = 0;
    cols = 0;
    d_data = nullptr;
}

Matrix::Matrix(int r, int c) {
    rows = r;
    cols = c;
    CUDA_CHECK(hipMalloc(&d_data, rows * cols * sizeof(float)));
}

Matrix::~Matrix() {
    if (d_data) hipFree(d_data);
}

// Copies data from host pointer to device memory
void Matrix::copyFromHost(const float* h_data) {
    CUDA_CHECK(hipMemcpy(d_data, h_data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
}

// Copies data from device pointer
void Matrix::copyFromDevice(const float* d_data_) {
    CUDA_CHECK(hipMemcpy(d_data, d_data_, rows * cols * sizeof(float), hipMemcpyDeviceToDevice));
}

// Copies data from device memory to host pointer
void Matrix::copyToHost(float* h_data) const {
    CUDA_CHECK(hipMemcpy(h_data, d_data, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
}

Vector::Vector() {
    n_elements = 0;
    d_data = nullptr;
}

Vector::Vector(int n) : n_elements(n) {
    CUDA_CHECK(hipMalloc(&d_data, n_elements * sizeof(float)));
}

Vector::~Vector() {
    if (d_data) hipFree(d_data);
}

// Copies data from host pointer to device memory
void Vector::copyFromHost(const float* h_data) {
    CUDA_CHECK(hipMemcpy(d_data, h_data, n_elements * sizeof(float), hipMemcpyHostToDevice));
}

// Copies data from device pointer
void Vector::copyFromDevice(const float* d_data_) {
    CUDA_CHECK(hipMemcpy(d_data, d_data_, n_elements * sizeof(float), hipMemcpyDeviceToDevice));
}

// Copies data from device memory to host pointer
void Vector::copyToHost(float* h_data) const {
    CUDA_CHECK(hipMemcpy(h_data, d_data, n_elements * sizeof(float), hipMemcpyDeviceToHost));
}

// -----------------------------
// CUDA Kernels for custom operations
// -----------------------------

// Kernel to perform vector difference
__global__ void subKernel(float* out, const float* v, const float* w, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) { 
        out[idx] = v[idx] - w[idx]; 
    }
}

// Kernel to add two vectors together
__global__ void addKernel(float* out, const float* v, const float* w, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) { 
        out[idx] = v[idx] + w[idx]; 
    }
}

// Kernel to scale a matrix in-place
__global__
void scaleMatrixKernel(float* data, float s, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = rows * cols;
    if (idx < total) {
        data[idx] *= s;
    }
}

// Kernel to scale a vector in-place
__global__
void scaleVectorKernel(float* data, float s, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] *= s;
    }
}

// Kernel for left diagonal matrix multiplication: out(i,j) = d[i] * A(i,j)
__global__
void leftDiagMatMulKernel(const float* d_vec, const float* A, float* out, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        out[col * rows + row] = d_vec[row] * A[col * rows + row];
    }
}

// Kernel for right diagonal matrix multiplication: out(i,j) = A(i,j) * d[j]
__global__
void rightDiagMatMulKernel(const float* A, const float* d_vec, float* out, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        out[col * rows + row] = d_vec[col] * A[col * rows + row];
    }
}

// Kernel to compute max absolute value per block for infinity norm reduction.
__global__
void reduceAbsMaxKernel(const float* in, float* blockMax, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    float myMax = 0.0f;
    if (i < n) {
        myMax = fabsf(in[i]);
    }
    sdata[tid] = myMax;
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s && i + s < n) {
            float other = sdata[tid + s];
            if (other > sdata[tid]) sdata[tid] = other;
        }
        __syncthreads();
    }
    // Write result for this block to global memory
    if (tid == 0) {
        blockMax[blockIdx.x] = sdata[0];
    }
}

// Custom kernel for computing A = A + alpha * I (computation performed in place)
// n is the number of rows of A.
__global__
void addScalarMatrixKernel(float* A, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        A[n * i + i] += alpha;
    }
}

// CUDA kernel to initialize a scaled diagonal matrix
// result = s * D, where D = diag(d)
__global__ void diagonal_kernel(float* d_matrix, const float* d_d, float s, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        d_matrix[row + col * n] = (row == col) ? d_d[row] * s : 0.0f;
    }
}

__global__ void diagonal_kernel(float* d_matrix, float s, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        d_matrix[row + col * n] = (row == col) ? s : 0.0f;
    }
}

__global__ void clipVectorKernel(float* v, const float* l, const float* u, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float v_i = v[i];
        float l_i = l[i];
        float u_i = u[i];
        if (v_i < l_i) {v[i] = l_i;}
        else {
            if (v_i > u_i) {v[i] = u_i;}
        }
    }
}

// -----------------------------
// Member function implementations
// -----------------------------

void Matrix::transpose(hipblasHandle_t handle, Matrix &dst) {
    if (dst.rows != cols || dst.cols != rows) {
        std::cerr << "Function transpose. Destination matrix dimensions don't match." << std::endl;
        exit(EXIT_FAILURE);
    }

    const float alpha = 1.0f, beta = 0.0f;
    hipblasSgeam(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,  // Transpose A, do not transpose B
        cols, rows,                // Dimensions of transposed matrix
        &alpha,
        d_data, rows,  // Input matrix (column-major)
        &beta,
        nullptr, cols,  // No second matrix (beta = 0)
        dst.d_data, cols  // Output matrix (column-major)
    );
}

void Matrix::scale(float s) {
    int total = rows * cols;
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    scaleMatrixKernel<<<gridSize, blockSize>>>(d_data, s, rows, cols);
    CUDA_CHECK(hipGetLastError());
}

void Matrix::addScalarMatrix(float alpha) {
    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;
    addScalarMatrixKernel<<<gridSize, blockSize>>>(d_data, alpha, rows);
    CUDA_CHECK(hipGetLastError());
}

void Vector::scale(float s) {
    // Copy original data to result
    int blockSize = 256;
    int gridSize = (n_elements + blockSize - 1) / blockSize;
    scaleVectorKernel<<<gridSize, blockSize>>>(d_data, s, n_elements);
    CUDA_CHECK(hipGetLastError());
}

// -----------------------------
// Free Functions for Linear Algebra
// -----------------------------

// Initialization of a matrix to the identity, using custom kernels.
void eye(Matrix &dst) {
    if (dst.rows != dst.cols) {
        std::cerr << "Function eye. Matrix is not square." << std::endl;
        exit(EXIT_FAILURE);
    }
    int n = dst.rows;
    // Define CUDA grid/block dimensions
    dim3 blockDim(16, 16);  // 16x16 threads per block
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
    // Launch kernel
    diagonal_kernel<<<gridDim, blockDim>>>(dst.d_data, 1, n);
    // Check for errors
    CUDA_CHECK(hipGetLastError());
}

// Vector sum using custom kernels
void vecsum(Vector &dst, const Vector &v, const Vector &w) {
    if (v.n_elements != w.n_elements || dst.n_elements != v.n_elements) {
        std::cerr << "Function vecsum. Vectors dimensions mismatch." << std::endl;
        exit(EXIT_FAILURE);
    }
    int blockSize = 256;
    int gridSize = (v.n_elements + blockSize - 1) / blockSize;
    addKernel<<<gridSize, blockSize>>>(dst.d_data, v.d_data, w.d_data, v.n_elements);
    CUDA_CHECK(hipGetLastError());
}

// Vector difference using custom kernels: result = v - w
void vecdiff(Vector &dst, const Vector &v, const Vector &w) {
    if (v.n_elements != w.n_elements || dst.n_elements != v.n_elements) {
        std::cerr << "Function vecdiff. Vectors dimensions mismatch." << std::endl;
        exit(EXIT_FAILURE);
    }
    Vector result(v.n_elements);
    int blockSize = 256;
    int gridSize = (dst.n_elements + blockSize - 1) / blockSize;
    subKernel<<<gridSize, blockSize>>>(dst.d_data, v.d_data, w.d_data, dst.n_elements);
    CUDA_CHECK(hipGetLastError());
}

// Matrix multiplication using cuBLAS: C = A * B
void matmul(hipblasHandle_t handle, Matrix &dst, const Matrix &A, const Matrix &B) {
    if (A.cols != B.rows) {
        std::cerr << "Function matmul. Matrix dimensions mismatch for multiplication." << std::endl;
        exit(EXIT_FAILURE);
    }
    if (A.rows != dst.rows || B.cols != dst.cols) {
        std::cerr << "Function matmul. Destination matrix is not of right dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }
    const float alpha = 1.0f, beta = 0.0f;
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             A.rows, B.cols, B.rows,
                             &alpha,
                             A.d_data, A.rows,
                             B.d_data, B.rows,
                             &beta,
                             dst.d_data, dst.rows));
}

// Clipping operations applied to a vector, through custom kernels
void clip(float* v, const Vector &l, const Vector &u) {
    if (l.n_elements != u.n_elements) {
        std::cerr << "Function clip(v, l, u).\nVectors dimensions mismatch." << std::endl;
        exit(EXIT_FAILURE);
    }
    int blockSize = 256;
    int gridSize = (l.n_elements + blockSize - 1) / blockSize;
    clipVectorKernel<<<gridSize, blockSize>>>(v, l.d_data, u.d_data, l.n_elements);
    CUDA_CHECK(hipGetLastError());
}

void diag(Matrix &dst, const Vector &d, float s) {
    if (dst.rows != d.n_elements || dst.cols != d.n_elements) {
        std::cerr << "Function diag. Matrix is not n x n." << std::endl;
        exit(EXIT_FAILURE);
    }
    int n = d.n_elements;

    // Define CUDA grid/block dimensions
    dim3 blockDim(16, 16);  // 16x16 threads per block
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    diagonal_kernel<<<gridDim, blockDim>>>(dst.d_data, d.d_data, s, n);

    // Check for errors
    CUDA_CHECK(hipGetLastError());
}

void matmul_scale(hipblasHandle_t handle, Matrix &dst, const Matrix &A, const Matrix &B, float s) {
    if (A.cols != B.rows) {
        std::cerr << "Matrix dimensions mismatch for multiplication." << std::endl;
        exit(EXIT_FAILURE);
    }
    if (A.rows != dst.rows || B.cols != dst.cols) {
        std::cerr << "Function matmul_scale. Destination matrix is not of right dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }
    const float alpha = s, beta = 0.0f;
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             A.rows, B.cols, B.rows,
                             &alpha,
                             A.d_data, A.rows,
                             B.d_data, B.rows,
                             &beta,
                             dst.d_data, dst.rows));
}

void matmul_scale_add(hipblasHandle_t handle, Matrix &dst, const Matrix &A, const Matrix &B, float s, const Matrix &C) {
    if (A.cols != B.rows or C.rows != A.rows or C.cols != B.cols) {
        std::cerr << "Matrix dimensions mismatch for multiplication and addition." << std::endl;
        exit(EXIT_FAILURE);
    }
    if (A.rows != dst.rows || B.cols != dst.cols) {
        std::cerr << "Function matmul_scale_add. Destination matrix is not of right dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate a new matrix
    int m = C.rows;
    int n = C.cols;
    CUDA_CHECK(hipMemcpy(dst.d_data, C.d_data, m * n * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Perform the computation D = D + s * AB
    const float alpha = s, beta = 1;
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             A.rows, B.cols, B.rows,
                             &alpha,
                             A.d_data, A.rows,
                             B.d_data, B.rows,
                             &beta,
                             dst.d_data, dst.rows));
}

void matvecmul(hipblasHandle_t handle, Vector &dst, const Matrix &W, const Vector &y) {
    if (W.cols != y.n_elements || W.rows != dst.n_elements) {
        std::cerr << "Function matvecmul. Dimension mismatch in matrix vector product." << std::endl;
        exit(EXIT_FAILURE);
    }
    const float alpha = 1.0f, beta = 0.0f;
    // Using cuBLAS for matrix-vector multiplication: result = W * y.
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_N,
                             W.rows, W.cols,
                             &alpha,
                             W.d_data, W.rows,
                             y.d_data, 1,
                             &beta,
                             dst.d_data, 1));
}

void matvecmul(hipblasHandle_t handle, float* dst, const Matrix &W, const Vector &y) {
    if (W.cols != y.n_elements) {
        std::cerr << "Function matvecmul. Dimension mismatch in matrix vector product." << std::endl;
        exit(EXIT_FAILURE);
    }
    const float alpha = 1.0f, beta = 0.0f;
    // Using cuBLAS for matrix-vector multiplication: result = W * y.
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_N,
                             W.rows, W.cols,
                             &alpha,
                             W.d_data, W.rows,
                             y.d_data, 1,
                             &beta,
                             dst, 1));
}

// Affine transformation: computes Wy + b.
void affine_transformation(hipblasHandle_t handle, Vector &dst, const Matrix &W, const Vector &y, const Vector &b) {
    if (W.cols != y.n_elements || W.rows != b.n_elements || W.rows != dst.n_elements) {
        std::cerr << "Function affine_transformation. Dimension mismatch in affine_transformation." << std::endl;
        exit(EXIT_FAILURE);
    }
    const float alpha = 1.0f, beta = 0.0f;
    //Vector intermediate(dst.n_elements);
    // Using cuBLAS for matrix-vector multiplication: y = W * y.
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_N,
                             W.rows, W.cols,
                             &alpha,
                             W.d_data, W.rows,
                             y.d_data, 1,
                             &beta,
                             dst.d_data, 1));
    // Add bias b: y = y + b
    int blockSize = 256;
    int gridSize = (y.n_elements + blockSize - 1) / blockSize;
    addKernel<<<gridSize, blockSize>>>(dst.d_data, dst.d_data, b.d_data, dst.n_elements);
    CUDA_CHECK(hipGetLastError());
}

// Left diagonal matrix multiplication: each row i of A is scaled by d[i].
void left_diag_matmul(Matrix &dst, const Vector &d, const Matrix &A) {
    if (d.n_elements != A.rows || dst.rows != A.rows || dst.cols != A.cols) {
        std::cerr << "Function left_diag_matmul. Dimension mismatch in left_diag_matmul." << std::endl;
        exit(EXIT_FAILURE);
    }
    dim3 blockDim(16, 16);
    dim3 gridDim((A.cols + blockDim.x - 1) / blockDim.x, (A.rows + blockDim.y - 1) / blockDim.y);
    leftDiagMatMulKernel<<<gridDim, blockDim>>>(d.d_data, A.d_data, dst.d_data, A.rows, A.cols);
    CUDA_CHECK(hipGetLastError());
}

void right_diag_matmul(Matrix &dst, const Matrix &A, const Vector &d) {
    if (d.n_elements != A.cols || dst.rows != A.rows || dst.cols != A.cols) {
        std::cerr << "Function right_diag_matmul. Dimension mismatch in right_diag_matmul." << std::endl;
        exit(EXIT_FAILURE);
    }
    dim3 blockDim(16, 16);
    dim3 gridDim((A.cols + blockDim.x - 1) / blockDim.x, (A.rows + blockDim.y - 1) / blockDim.y);
    rightDiagMatMulKernel<<<gridDim, blockDim>>>(A.d_data, d.d_data, dst.d_data, A.rows, A.cols);
    CUDA_CHECK(hipGetLastError());
}

// Compute the infinity norm (maximum absolute value) of a vector using a reduction kernel.
float infinity_norm(const Vector &v) {
    int n = v.n_elements;
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    // Allocate temporary array for block maximums.
    float* d_blockMax;
    CUDA_CHECK(hipMalloc(&d_blockMax, gridSize * sizeof(float)));
    size_t sharedMemSize = blockSize * sizeof(float);
    reduceAbsMaxKernel<<<gridSize, blockSize, sharedMemSize>>>(v.d_data, d_blockMax, n);
    CUDA_CHECK(hipGetLastError());

    // Copy block results to host and reduce on CPU.
    float* h_blockMax = new float[gridSize];
    CUDA_CHECK(hipMemcpy(h_blockMax, d_blockMax, gridSize * sizeof(float), hipMemcpyDeviceToHost));
    float maxVal = 0.0f;
    for (int i = 0; i < gridSize; ++i) {
        if (h_blockMax[i] > maxVal)
            maxVal = h_blockMax[i];
    }
    delete[] h_blockMax;
    hipFree(d_blockMax);
    return maxVal;
}
